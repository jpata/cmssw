#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "HeterogeneousCore/CUDAUtilities/interface/cudaCheck.h"
#include "HeterogeneousCore/CUDAUtilities/interface/SimpleVector.h"

static const float cristalPhiEtaMaxSize_ = 0.04f;

__global__ void kernel_track_cluster_dr(
  const float* track_pt,
  const float* track_eta,
  const float* track_phi,
  size_t num_tracks,
  const float* rechit_eta,
  const float* rechit_phi,
  const int* rechit_clusteridx,
  size_t num_rechits,
  int* out_track_clusteridx)
{
  //process each track in a different CUDA thread
  for (size_t i=blockDim.x*blockIdx.x+threadIdx.x; i<num_tracks; i+=blockDim.x*gridDim.x) {

    const float range = cristalPhiEtaMaxSize_ * (2.0f + 1.0f / std::min(1.0f, track_pt[i] / 2.0f));
    printf("A track=%d pt=%.2f range=%.2f\n", (int)i, track_pt[i], range);

    // find all rechits in box around track with track_eta+-range, track_phi+-range
    cms::cuda::SimpleVector<int> rechit_inds;
    for (size_t j=0; j<num_rechits; j++) {
      const float deta = std::abs(track_eta[i] - rechit_eta[j]);
      const float dphi = std::abs(track_phi[i] - rechit_phi[j]);
      const bool match = (deta<range && dphi<range);
      if (match) {
        rechit_inds.push_back_unsafe(j);
      }
    }

    // loop over clusters associated to each found rechit
    for (const auto rh_idx : rechit_inds) {
    }
  } //loop over tracks
  

}


 
void track_cluster_dr(
  const float* track_pt,
  const float* track_eta,
  const float* track_phi,
  size_t num_tracks,
  const float* rechit_eta,
  const float* rechit_phi,
  const int* rechit_clusteridx,
  size_t num_rechits,
  int* out_track_clusteridx)
{
  kernel_track_cluster_dr<<<1, 1>>>(track_pt, track_eta, track_phi, num_tracks, rechit_eta, rechit_phi, rechit_clusteridx, num_rechits, out_track_clusteridx);
  cudaCheck(hipDeviceSynchronize());
  cudaCheck(hipGetLastError());
}

